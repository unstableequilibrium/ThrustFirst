#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/sort.h>
#include <thrust/copy.h>
#include <thrust/detail/type_traits.h>
#include <algorithm>
#include <cstdlib>
#include <time.h>
#include <limits.h>
#include <fstream>
#include <string>
#include <iostream>
#include <sstream>


#ifdef _WIN32
#include <windows.h>
#elif __APPLE__
#include <sys/stat.h>
#include <stdlib.h>
#include <unistd.h>
#endif

using namespace std;
using namespace thrust;

struct values_vec
{
	float X;
	float Y;
	float Z;
	float Vr;
	float Vs;
	float Vt;
};

void FirstReductionStep(vector<values_vec> &hostData, thrust::host_vector<int> &hostSortID, thrust::host_vector<int> &hostKeyData,
	vector<values_vec> &intermediate, vector<int> &intermediateKey, bool isBinary,
	ofstream &finalOut, int &reducedRecs, bool isOptionA = false, string pathInter = "")
{
	ofstream intermediateOut;
	vector<int> dataIds, keyIds;
	if (isOptionA){
#ifdef _WIN32
		string fileName = pathInter + string("\\intermediate1");
#elif __APPLE__
		string fileName = pathInter + string("/intermediate1");
#endif
		if (isBinary){
			fileName += string(".bin");
			intermediateOut.open(fileName.c_str(), ios::binary);
		}
		else{
			fileName += string(".txt");
			intermediateOut.open(fileName.c_str());
		}
	}
	reducedRecs = 0;
	// checking in treshold values
	int N = hostData.size();
	for (int j = 0; j < N; j++){
		// Find number of records with same key
		int sameKeys = 1;
		while (j < N - 1){
			if ((hostKeyData[j] >> 3) == (hostKeyData[j + 1] >> 3))j++, sameKeys++;
			else break;
		}

		int curID;
		if (sameKeys == 1 && isOptionA){
			curID = hostSortID[j];
			if (!isBinary){
				finalOut << hostData[curID].X << " " << hostData[curID].Y << " " << hostData[curID].Z << " ";
				finalOut << hostKeyData[j] << " ";
				finalOut << hostData[curID].Vr << " " << hostData[curID].Vs << " " << hostData[curID].Vt << std::endl;
			}
			continue;
		}

		// calculate mean values of current same key records
		values_vec meanVals;
		meanVals.Vr = meanVals.Vs = meanVals.Vt = 0;
		for (int i = j - sameKeys + 1; i <= j; i++){
			curID = hostSortID[i];
			meanVals.Vr += hostData[curID].Vr;
			meanVals.Vs += hostData[curID].Vs;
			meanVals.Vt += hostData[curID].Vt;
		}
		meanVals.Vr /= sameKeys;
		meanVals.Vs /= sameKeys;
		meanVals.Vt /= sameKeys;

		// calculate standard deviations 
		values_vec stdVals;
		stdVals.Vr = stdVals.Vs = stdVals.Vt = 0;
		for (int i = j - sameKeys + 1; i <= j; i++){
			curID = hostSortID[i];
			stdVals.Vr += (hostData[curID].Vr - meanVals.Vr) * (hostData[curID].Vr - meanVals.Vr);
			stdVals.Vs += (hostData[curID].Vs - meanVals.Vs) * (hostData[curID].Vs - meanVals.Vs);
			stdVals.Vt += (hostData[curID].Vt - meanVals.Vt) * (hostData[curID].Vt - meanVals.Vt);
		}
		stdVals.Vr = sqrtf(stdVals.Vr / sameKeys);
		stdVals.Vs = sqrtf(stdVals.Vs / sameKeys);
		stdVals.Vt = sqrtf(stdVals.Vt / sameKeys);

		// checking treshold
		bool inTresh = true;
		for (int i = j - sameKeys + 1; i <= j; i++){
			curID = hostSortID[i];
			int z1 = (int) (fabsf(hostData[curID].Vr - meanVals.Vr) / stdVals.Vr + 0.5);
			int z2 = (int) (fabsf(hostData[curID].Vs - meanVals.Vs) / stdVals.Vs + 0.5);
			int z3 = (int) (fabsf(hostData[curID].Vt - meanVals.Vt) / stdVals.Vt + 0.5);
			if (z1 != 1 || z2 != 1 || z3 != 1){
				inTresh = false;
				break;
			}
		}
		if (inTresh){
			// write to intermediate data
			int oldKey = hostKeyData[j - sameKeys + 1];
			int curID = hostSortID[j - sameKeys + 1];
			meanVals.X = hostData[curID].X; meanVals.Y = hostData[curID].Y; meanVals.Z = hostData[curID].Z;
			intermediate.push_back(meanVals);
			intermediateKey.push_back(oldKey);

			reducedRecs += sameKeys;
			if (isOptionA){
				if (!isBinary){
					intermediateOut << intermediate.back().X << " " << intermediate.back().Y << " " << intermediate.back().Z << " ";
					intermediateOut << (intermediateKey.back() >> 3) << " ";
					intermediateOut << intermediate.back().Vr << " " << intermediate.back().Vs << " " << intermediate.back().Vt << " ";
					intermediateOut << intermediateKey.back() << endl;
				}
			}
		}
		else if (isOptionA){
			// write to final file
			if (!isBinary){
				ostringstream out;
				for (int i = j - sameKeys + 1; i <= j; i++){
					curID = hostSortID[i];
					values_vec tmp = hostData[curID];
					out << tmp.X << " " << tmp.Y << " " << tmp.Z << " ";
					out << hostKeyData[i] << " ";
					out << tmp.Vr << " " << tmp.Vs << " " << tmp.Vt << std::endl;
				}
				finalOut << out.str();
			}
			else{
				for (int i = j - sameKeys + 1; i <= j; i++){
					dataIds.push_back(hostSortID[i]);
					keyIds.push_back(hostKeyData[i]);
				}
			}
		}
	}

	if (isOptionA){
		// write to binary files
		if (isBinary){
			int finalSz = dataIds.size();
			int reduceSz = finalSz >> 3;
			char *Buffer = new char[28 * reduceSz];
			float *ptr = (float *) Buffer;
			int counter = 0;
			vector<int>::iterator itK = keyIds.begin();
			for (int i = 0; i < finalSz; i++){
				int curID = dataIds[i];
				values_vec tmp = hostData[curID];
				*(ptr++) = tmp.X; *(ptr++) = tmp.Y; *(ptr++) = tmp.Z;
				*(ptr++) = *((float *) (&(*(itK++))));
				*(ptr++) = tmp.Vr; *(ptr++) = tmp.Vs; *(ptr++) = tmp.Vt;
				counter++;
				if (counter % reduceSz == 0){
					finalOut.write(Buffer, 28 * reduceSz);
					ptr = (float *) Buffer;
				}
			}
			int remain = counter % reduceSz;
			if (remain)finalOut.write(Buffer, 28 * remain);

			delete [] Buffer;

			// Intermediate
			finalSz = intermediate.size();
			Buffer = new char[32 * finalSz];
			ptr = (float *) Buffer;
			for (int i = 0; i < finalSz; i++){
				values_vec tmp = intermediate[i];
				*(ptr++) = tmp.X; *(ptr++) = tmp.Y; *(ptr++) = tmp.Z;
				int newKey = intermediateKey[i] >> 3;
				*(ptr++) = *((float *) (&newKey));
				*(ptr++) = tmp.Vr; *(ptr++) = tmp.Vs; *(ptr++) = tmp.Vt;
				*(ptr++) = *((float *) (&intermediateKey[i]));
			}
			intermediateOut.write(Buffer, 32 * finalSz);
			delete [] Buffer;
		}
		intermediateOut.close();
	}
}

void NextReductionStep(int step, vector<values_vec> &hostData, thrust::host_vector<int> &hostKeyData,
	vector<values_vec> &intermediate, vector<int> &intermediateKey, bool isBinary,
	ofstream &finalOut, int &reducedRecs, bool isOptionA = false, string pathInter = "")
{
	reducedRecs = 0;
	int N = hostData.size();
	if (N == 1 && isOptionA){
		if (isBinary){
			finalOut << hostData[0].X << hostData[0].Y << hostData[0].Z;
			finalOut << hostKeyData[0] << hostData[0].Vr << hostData[0].Vs << hostData[0].Vt;
		}
		else {
			finalOut << hostData[0].X << " " << hostData[0].Y << " " << hostData[0].Z << " ";
			finalOut << hostKeyData[0] << " ";
			finalOut << hostData[0].Vr << " " << hostData[0].Vs << " " << hostData[0].Vt << std::endl;
		}
		return;
	}

	ofstream intermediateOut;
	vector<int> finalIds;
	if (isOptionA){
		ostringstream filename;
#ifdef _WIN32
		filename << pathInter << "\\intermediate" << (step + 1);
#elif __APPLE__
		filename << pathInter << "/intermediate" << (step + 1);
#endif
		if (isBinary){
			filename << ".bin";
			intermediateOut.open(filename.str().c_str(), ios::binary);
		}
		else{
			filename << ".txt";
			intermediateOut.open(filename.str().c_str());
		}
	}

	// checking in treshold values
	int shift = 3 * step + 3;
	for (int j = 0; j < N; j++){
		// Find number of records with same key
		int sameKeys = 1;
		while (j < N - 1){
			if ((hostKeyData[j] >> shift) == (hostKeyData[j + 1] >> shift))j++, sameKeys++;
			else break;
		}

		if (sameKeys == 1 && isOptionA){
			if (!isBinary){
				finalOut << hostData[j].X << " " << hostData[j].Y << " " << hostData[j].Z << " ";
				finalOut << hostKeyData[j] << " ";
				finalOut << hostData[j].Vr << " " << hostData[j].Vs << " " << hostData[j].Vt << std::endl;
			}
			else{
				finalIds.push_back(j);
			}
			continue;
		}

		// calculate mean values of current same key records
		values_vec meanVals;
		meanVals.Vr = meanVals.Vs = meanVals.Vt = 0;
		for (int i = j - sameKeys + 1; i <= j; i++){
			meanVals.Vr += hostData[i].Vr;
			meanVals.Vs += hostData[i].Vs;
			meanVals.Vt += hostData[i].Vt;
		}
		meanVals.Vr /= sameKeys;
		meanVals.Vs /= sameKeys;
		meanVals.Vt /= sameKeys;

		// calculate standard deviations 
		values_vec stdVals;
		stdVals.Vr = stdVals.Vs = stdVals.Vt = 0;
		for (int i = j - sameKeys + 1; i <= j; i++){
			stdVals.Vr += (hostData[i].Vr - meanVals.Vr) * (hostData[i].Vr - meanVals.Vr);
			stdVals.Vs += (hostData[i].Vs - meanVals.Vs) * (hostData[i].Vs - meanVals.Vs);
			stdVals.Vt += (hostData[i].Vt - meanVals.Vt) * (hostData[i].Vt - meanVals.Vt);
		}
		stdVals.Vr = sqrtf(stdVals.Vr / sameKeys);
		stdVals.Vs = sqrtf(stdVals.Vs / sameKeys);
		stdVals.Vt = sqrtf(stdVals.Vt / sameKeys);

		// checking treshold
		bool inTresh = true;
		for (int i = j - sameKeys + 1; i <= j; i++){
			int z1 = (int) (fabsf(hostData[i].Vr - meanVals.Vr) / stdVals.Vr + 0.5);
			int z2 = (int) (fabsf(hostData[i].Vs - meanVals.Vs) / stdVals.Vs + 0.5);
			int z3 = (int) (fabsf(hostData[i].Vt - meanVals.Vt) / stdVals.Vt + 0.5);
			if (z1 != 1 || z2 != 1 || z3 != 1){
				inTresh = false;
				break;
			}
		}

		if (inTresh){
			// write to intermediate data
			int oldKey = hostKeyData[j - sameKeys + 1];
			int curID = j - sameKeys + 1;
			meanVals.X = hostData[curID].X; meanVals.Y = hostData[curID].Y; meanVals.Z = hostData[curID].Z;
			intermediate.push_back(meanVals);
			intermediateKey.push_back(oldKey);
			reducedRecs += sameKeys;
			if (isOptionA){
				if (!isBinary){
					intermediateOut << intermediate.back().X << " " << intermediate.back().Y << " " << intermediate.back().Z << " ";
					intermediateOut << (intermediateKey.back() >> 3) << " ";
					intermediateOut << intermediate.back().Vr << " " << intermediate.back().Vs << " " << intermediate.back().Vt << " ";
					intermediateOut << intermediateKey.back() << endl;
				}
			}
		}
		else if (isOptionA){
			if (!isBinary){
				ostringstream out;
				for (int i = j - sameKeys + 1; i <= j; i++){
					values_vec tmp = hostData[i];
					out << tmp.X << " " << tmp.Y << " " << tmp.Z << " ";
					out << hostKeyData[i] << " ";
					out << tmp.Vr << " " << tmp.Vs << " " << tmp.Vt << std::endl;
				}
				finalOut << out.str();
			}
			else{
				for (int i = j - sameKeys + 1; i <= j; i++){
					finalIds.push_back(i);
				}
			}
		}
	}
	if (isOptionA){
		// write to binary files
		if (isBinary){
			// Final
			int finalSz = finalIds.size();
			char *Buffer = new char[28 * finalSz];
			float *ptr = (float *) Buffer;
			for (int i = 0; i < finalSz; i++){
				int curID = finalIds[i];
				values_vec tmp = hostData[curID];
				*(ptr++) = tmp.X; *(ptr++) = tmp.Y; *(ptr++) = tmp.Z;
				*(ptr++) = *((float *) (&hostKeyData[curID]));
				*(ptr++) = tmp.Vr; *(ptr++) = tmp.Vs; *(ptr++) = tmp.Vt;
			}
			finalOut.write(Buffer, 28 * finalSz);
			delete [] Buffer;

			// Intermediate
			finalSz = intermediate.size();
			Buffer = new char[32 * finalSz];
			ptr = (float *) Buffer;
			for (int i = 0; i < finalSz; i++){
				values_vec tmp = intermediate[i];
				*(ptr++) = tmp.X; *(ptr++) = tmp.Y; *(ptr++) = tmp.Z;
				int newKey = intermediateKey[i] >> 3;
				*(ptr++) = *((float *) (&newKey));
				*(ptr++) = tmp.Vr; *(ptr++) = tmp.Vs; *(ptr++) = tmp.Vt;
				*(ptr++) = *((float *) (&intermediateKey[i]));
			}
			intermediateOut.write(Buffer, 32 * finalSz);
			delete [] Buffer;

		}
		intermediateOut.close();
	}
}

void Merge2SortedHalves(thrust::host_vector<int> &key, thrust::host_vector<int> &val, int begin, int end,
	thrust::host_vector<int> &keySorted, thrust::host_vector<int> &valSorted, int beginSorted)
{
	int j = 0, k = 0;
	int N = end - begin;
	int N2 = N >> 1;
	for (int i = N2; i < N; i++){
		if (j != N2){
			while (key[begin + i] > key[begin + j]){
				valSorted[beginSorted + k] = val[begin + j];
				keySorted[beginSorted + k] = key[begin + j];
				k++, j++;
				if (j == N2)break;
			}
		}
		valSorted[beginSorted + k] = val[begin + i];
		keySorted[beginSorted + k] = key[begin + i];
		k++;
	}
	while (j < N2){
		valSorted[beginSorted + k] = val[begin + j];
		keySorted[beginSorted + k] = key[begin + j];
		k++, j++;
	}
}

void OptimizedReadFile(int N, ifstream &input, char delim, vector<values_vec> &hostData, thrust::host_vector<int> &hostKeyData)
{
	char buf[256];
	vector<values_vec>::iterator it = hostData.begin();
	thrust::host_vector<int>::iterator itK = hostKeyData.begin();
	for (int i = 0; i < N; i++){
		input.getline(buf, 256);
		char *ch = buf;
		(*it).X = atof(ch);
		while (*ch++ != delim);
		(*it).Y = atof(ch);
		while (*ch++ != delim);
		(*it).Z = atof(ch);
		while (*ch++ != delim);
		while (*ch++ != delim);
		while (*ch++ != delim);
		while (*ch++ != delim);
		*itK = atoi(ch);
		while (*ch++ != delim);
		(*it).Vr = atof(ch);
		while (*ch++ != delim);
		(*it).Vs = atof(ch);
		while (*ch++ != delim);
		(*it).Vt = atof(ch);
		it++;
		itK++;
	}
	input.close();
}

void ReductionOfFile(int x, ifstream &input, float &TotalTime, bool isOptionA, bool withGPU, bool isBinary, ofstream &finalOut, ofstream &timeOut, string pathInter = "")
{
	int N = 1 << (3 * x);
	int N2 = N >> 1, N4 = N >> 2;
	clock_t cpu_time = clock();
	float allocationTime, readTime;
	// allocate host data of values on CPU
	std::vector<values_vec> hostData(N);
	thrust::host_vector<int> hostKeyData(N);
	thrust::host_vector<int> hostSortID(N);

	float totalSort = 0;
	if (withGPU){
		// additional memory for merging
		thrust::host_vector<int> hostKeyData1(N);
		thrust::host_vector<int> hostSortID1(N);

		// allocate thrust::device data on GPU
		thrust::device_vector<int> devKeyData(N2);
		thrust::device_vector<int> devSortID(N2);
		allocationTime = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "allocation Data time = " << allocationTime << std::endl;
		if (isOptionA)timeOut << "allocation Data time = " << allocationTime << std::endl;

		// Reading original file 
		//std::cout<<"start reading data from input to host \n";
		cpu_time = clock();

		// Optimized read original file which have N records with delimiter = '\t' 
		OptimizedReadFile(N, input, '\t', hostData, hostKeyData);
		// init vec of IDs
		for (int i = 0; i < N; i++)hostSortID[i] = i;

		readTime = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "reading file time = " << readTime << std::endl;
		if (isOptionA)timeOut << "reading file time = " << readTime << std::endl;

		// copy 1-st part host data to device GPU data 
		//std::cout<<"copy key data to GPU \n";
		cpu_time = clock();
		thrust::copy(hostKeyData.begin(), hostKeyData.begin() + N2, devKeyData.begin());
		thrust::copy(hostSortID.begin(), hostSortID.begin() + N2, devSortID.begin());
		float copyToGPU = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "copy keys to GPU time = " << copyToGPU << std::endl;
		if (isOptionA)timeOut << "copy keys to GPU time = " << copyToGPU << std::endl;

		// sort 1-st part key column device data
		//std::cout<<"sort key data on GPU \n";
		cpu_time = clock();
		thrust::sort_by_key(devKeyData.begin(), devKeyData.begin() + N4, devSortID.begin());
		thrust::sort_by_key(devKeyData.begin() + N4, devKeyData.begin() + N2, devSortID.begin() + N4);
		float sortTime = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "sort keys on GPU time = " << sortTime << std::endl;
		if (isOptionA)timeOut << "sort keys on GPU time = " << sortTime << std::endl;

		// copy data fron device to host
		//std::cout<<"copy key data from GPU to host \n";
		cpu_time = clock();
		thrust::copy(devKeyData.begin(), devKeyData.end(), hostKeyData.begin());
		thrust::copy(devSortID.begin(), devSortID.end(), hostSortID.begin());
		float copyToCPU = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "copy keys from GPU to CPU time = " << copyToCPU << std::endl;
		if (isOptionA)timeOut << "copy keys from GPU to CPU time = " << copyToCPU << std::endl;

		// copy 2-nd part host data to device GPU data 
		//std::cout<<"copy key data to GPU \n";
		cpu_time = clock();
		thrust::copy(hostKeyData.begin() + N2, hostKeyData.end(), devKeyData.begin());
		thrust::copy(hostSortID.begin() + N2, hostSortID.end(), devSortID.begin());
		float copyToGPU2 = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "copy keys to GPU time = " << copyToGPU << std::endl;
		if (isOptionA)timeOut << "copy keys to GPU time = " << copyToGPU << std::endl;

		// sort 2-nd part key column device data
		//std::cout<<"sort key data on GPU \n";
		cpu_time = clock();
		thrust::sort_by_key(devKeyData.begin(), devKeyData.begin() + N4, devSortID.begin());
		thrust::sort_by_key(devKeyData.begin() + N4, devKeyData.begin() + N2, devSortID.begin() + N4);
		float sortTime2 = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "sort keys on GPU time = " << sortTime << std::endl;
		if (isOptionA)timeOut << "sort keys on GPU time = " << sortTime << std::endl;

		// copy data fron device to host
		//std::cout<<"copy key data from GPU to host \n";
		cpu_time = clock();
		thrust::copy(devKeyData.begin(), devKeyData.end(), hostKeyData.begin() + N2);
		thrust::copy(devSortID.begin(), devSortID.end(), hostSortID.begin() + N2);
		float copyToCPU2 = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "copy keys from GPU to CPU time = " << copyToCPU << std::endl;
		if (isOptionA)timeOut << "copy keys from GPU to CPU time = " << copyToCPU << std::endl;

		// 2 Merging 4 sorted quarters from GPU into 2 sorted halves vector on CPU
		cpu_time = clock();
		Merge2SortedHalves(hostKeyData, hostSortID, 0, N2, hostKeyData1, hostSortID1, 0);
		Merge2SortedHalves(hostKeyData, hostSortID, N2, N, hostKeyData1, hostSortID1, N2);
		// Merging 2 halves
		Merge2SortedHalves(hostKeyData1, hostSortID1, 0, N, hostKeyData, hostSortID, 0);
		float mergeTime = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "Merge on CPU time = " << mergeTime << std::endl;
		if (isOptionA)timeOut << "Merge on CPU time = " << mergeTime << std::endl;

		totalSort += copyToCPU + copyToGPU + sortTime + copyToCPU2 + copyToGPU2 + sortTime2 + mergeTime;
		// clear unnecessary data
		hostKeyData1.clear();
		hostSortID1.clear();
	}
	else{
		allocationTime = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "allocation Data time = " << allocationTime << std::endl;
		if (isOptionA)timeOut << "allocation Data time = " << allocationTime << std::endl;

		// Reading original file 
		//std::cout<<"start reading data from input to host \n";
		cpu_time = clock();

		// Optimized read original file which have N records with delimiter = '\t' 
		OptimizedReadFile(N, input, '\t', hostData, hostKeyData);
		// init vec of IDs
		for (int i = 0; i < N; i++)hostSortID[i] = i;

		readTime = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "reading file time = " << readTime << std::endl;
		if (isOptionA)timeOut << "reading file time = " << readTime << std::endl;

		// sort keys 
		cpu_time = clock();
		thrust::sort_by_key(hostKeyData.begin(), hostKeyData.end(), hostSortID.begin());
		totalSort = float(clock() - cpu_time) / CLOCKS_PER_SEC;
		cout << "Sort on CPU time = " << totalSort << std::endl;
	}

	// First reduction step
	//std::cout<<"start 1-st reduction step \n";
	vector<values_vec> intermediate;
	vector<int> intermediateKey;
	vector<float> iterationTime;
	int reducedRecs = 0, finalRecs = 0;
	cpu_time = clock();
	FirstReductionStep(hostData, hostSortID, hostKeyData, intermediate, intermediateKey, isBinary, finalOut, reducedRecs, isOptionA, pathInter);
	iterationTime.push_back(float(clock() - cpu_time) / CLOCKS_PER_SEC);
	finalRecs += (hostKeyData.size() - reducedRecs);
	cout << "1-iteration time = " << iterationTime.back() << std::endl;
	cout << "After 1 iteration: " << intermediate.size() << " records to Intermediate step" << endl;
	cout << "                   " << (hostKeyData.size() - reducedRecs) << " records to Final output" << endl;
	if (isOptionA)timeOut << "1-iteration time = " << iterationTime.back() << std::endl;
	hostSortID.clear();

	// next intermediate reduction steps
	int k = 1;
	while (k < x && !intermediate.empty()){
		cpu_time = clock();
		hostData = intermediate;
		hostKeyData = intermediateKey;
		intermediate.clear();
		intermediateKey.clear();
		reducedRecs = 0;
		NextReductionStep(k, hostData, hostKeyData, intermediate, intermediateKey, isBinary, finalOut, reducedRecs, isOptionA, pathInter);
		iterationTime.push_back(float(clock() - cpu_time) / CLOCKS_PER_SEC);
		finalRecs += (hostKeyData.size() - reducedRecs);
		cout << (k + 1) << "-iteration time = " << iterationTime.back() << std::endl;
		cout << "After " << (k + 1) << "iteration: " << intermediate.size() << " records to Intermediate step" << endl;
		cout << "                   " << (hostKeyData.size() - reducedRecs) << " records to Final output" << endl;
		if (isOptionA)timeOut << (k + 1) << "-iteration time = " << iterationTime.back() << std::endl;
		k++;
	}
	cout << "Total Final records: " << finalRecs << endl;

	// Total iterations time
	float totalItTime = 0;
	for (int i = 0; i < iterationTime.size(); i++)totalItTime += iterationTime[i];
	// Total time
	TotalTime = allocationTime + readTime + totalSort + totalItTime;

	if (isOptionA){
		cout << "Total iterations time = " << totalItTime << std::endl;
		timeOut << "Total iterations time = " << totalItTime << std::endl;


		cout << "Total time = " << TotalTime << std::endl;
		timeOut << "Total time = " << TotalTime << std::endl;

		timeOut.close();
		finalOut.close();
	}
}

int main(int argc, char **argv)
{
	int x = 7;// degree of 8: N = 8^x
	int NumOfFiles = 1;// number of input files in directory or number of iteration for one file
	char pathToRoot[1024];// path to folder with input files
	ofstream totalTimeOut("totalTime.txt");
	float totalTime = 0;

	bool isOptionA = true;// if false will be Option B (only total time for same N files)
	bool withGPU = true;// if false without using GPU only CPU
	bool isBinary = true;// if false -> txt format
	// for option B file name of one file which will be iterated N times
	string inputSingleFile("input1.txt");

#ifdef _WIN32 // FOR Windows

	std::ofstream finalOut;
	std::ofstream timeOut;
	if (isOptionA){
		GetCurrentDirectory(1024, pathToRoot);
		string folderFinal = string(pathToRoot) + string("\\FINAL");
		string folderInter = string(pathToRoot) + string("\\INTERMEDIATE");
		string folderTime = string(pathToRoot) + string("\\TIME");
		CreateDirectory(folderFinal.c_str(), NULL);
		CreateDirectory(folderInter.c_str(), NULL);
		CreateDirectory(folderTime.c_str(), NULL);
		for (int i = 0; i < NumOfFiles; i++){
			ostringstream fileName;
			fileName << "input" << (i + 1) << ".txt";
			std::ifstream input(fileName.str().c_str());
			if (!input.is_open()){
				cout << "File " << fileName.str().c_str() << " not found." << endl;
				continue;
			}
			else {
				cout << "File " << fileName.str().c_str() << " is opened" << endl;
			}
			ostringstream finalFile;
			finalFile << folderFinal << "\\final" << (i + 1);
			if (isBinary){
				finalFile << ".bin";
				finalOut.open(finalFile.str().c_str(), ofstream::binary);
			}
			else {
				finalFile << ".txt";
				finalOut.open(finalFile.str().c_str());
			}
			ostringstream  timeFile;
			timeFile << folderTime << "\\time" << (i + 1) << ".txt";
			timeOut.open(timeFile.str().c_str());

			ostringstream folderName;
			folderName << folderInter << "\\" << (i + 1);
			CreateDirectory(folderName.str().c_str(), NULL);

			float time;
			ReductionOfFile(x, input, time, isOptionA, withGPU, isBinary, finalOut, timeOut, folderName.str());
			totalTime += time;
		}
	}
	else {// Option B
		for (int i = 0; i < NumOfFiles; i++){
			float time;
			std::ifstream input(inputSingleFile.c_str());
			if (!input.is_open()){
				cout << "File " << inputSingleFile.c_str() << " not found" << endl;
				continue;
			}
			else{
				cout << "File " << inputSingleFile.c_str() << " is opened" << endl;
			}
			ReductionOfFile(x, input, time, isOptionA, withGPU, isBinary, finalOut, timeOut);
			totalTime += time;
		}
	}

#elif __APPLE__ // FOR MAC OSX

	std::ofstream finalOut;
	std::ofstream timeOut;
	if (isOptionA){
		getcwd(pathToRoot, 1024);
		string folderFinal = string(pathToRoot) + string("/FINAL");
		string folderInter = string(pathToRoot) + string("/INTERMEDIATE");
		string folderTime = string(pathToRoot) + string("/TIME");
		mkdir(folderFinal.c_str(), 0777);
		mkdir(folderInter.c_str(), 0777);
		mkdir(folderTime.c_str(), 0777);
		for (int i = 0; i < NumOfFiles; i++){
			ostringstream fileName;
			fileName << "input" << (i + 1) << ".txt";
			std::ifstream input(fileName.str().c_str());
			if (!input.is_open()){
				cout << "File " << fileName.str().c_str() << " not found." << endl;
				continue;
			}
			else {
				cout << "File " << fileName.str().c_str() << " is opened" << endl;
			}
			ostringstream finalFile;
			finalFile << folderFinal << "/final" << (i + 1);
			if (isBinary){
				finalFile << ".bin";
				finalOut.open(finalFile.str().c_str(), , ofstream::binary);
			}
			else {
				finalFile << ".txt";
				finalOut.open(finalFile.str().c_str());
			}

			ostringstream  timeFile;
			timeFile << folderTime << "/time" << (i + 1) << ".txt";
			timeOut.open(timeFile.str().c_str());

			ostringstream folderName;
			folderName << folderInter << "/" << (i + 1);
			mkdir(folderName.str().c_str(), 0777);
			float time;
			ReductionOfFile(x, input, time, isOptionA, withGPU, isBinary, finalOut, timeOut, folderName.str());
			totalTime += time;
		}
	}
	else {// Option B
		for (int i = 0; i < NumOfFiles; i++){
			float time;
			std::ifstream input(inputSingleFile.c_str());
			if (!input.is_open()){
				cout << "File " << inputSingleFile.c_str() << " not found" << endl;
				continue;
			}
			else{
				cout << "File " << inputSingleFile.c_str() << " is opened" << endl;
			}
			ReductionOfFile(x, input, time, isOptionA, withGPU, isBinary, finalOut, timeOut);
			totalTime += time;
		}
	}


#endif

	cout << "Total time = " << totalTime << endl;
	totalTimeOut << totalTime;
	totalTimeOut.close();

	return 0;
}